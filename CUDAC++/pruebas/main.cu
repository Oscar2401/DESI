#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>
#include <chrono>

typedef std::chrono::high_resolution_clock Clock;

using namespace std;

 
/*
=================================================
        Prototipos de fuciones CPU
=================================================
*/

void suma_cpu(float *, float *, float *, int);


/*
=================================================
        Prototipos de fuciones GPU
=================================================
*/

__global__
void suma_gpu(float *, float *, float*, int);

int main(int argc, char *argv[]){
    int N = 1e6, i;
    cout << N << endl;
    float *a, *b,*c, *C_gpu;
    a = (float*)malloc(N*sizeof(float));
    b = (float*)malloc(N*sizeof(float));
    c = (float*)malloc(N*sizeof(float));
    hipMallocManaged(&C_gpu,N*sizeof(float));
    
    for(i = 0; i<n; i++){
        *(a+i) = 1.0;
        *(b+i) = 4.0;
    }

    auto cpu_start = Clock::now();
    suma_cpu(a,b,c,N);
    auto cpu_end = Clock::now();

    auto gpu_start = Clock::now();
    suma_gpu<<<1,1>>>(a,b,C_gpu,N);
    hipDeviceSynchronize();
    auto gpu_end = Clock::now();

    for(i=0; i<10;i++){
        cout << *(c+i) << "\t" << *(C_gpu + i) << endl;
    }

    free(a);
    free(b);
    free(c);
    hipFree(C_gpu);

    return 0;
}


void suma_cpu(float *a, float *b, float *c, int n){

    for(int i = 0; i<n ; i++){
        *(c+i) = *(a+i) + *(b+i);
    }

}



__global__
void suma_gpu(float *a, float *b, float* c, int n){

    for(int i = 0; i < n; i++){
        *(c+i) = *(a+i) + *(b+i);
    }

}