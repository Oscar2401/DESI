// nvcc simple.cu -o par_s.out && ./par_s.out data_5K.dat rand0_5K.dat 5000 30 50

// For dynamic parallelism
// nvcc -arch=sm_35 -rdc=true dynamic.cu -lcudadevrt -o par_d.out && ./par_d.out data_5K.dat rand0_5K.dat 5000 30 50

#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <string.h>
#include <time.h>
#include <math.h>

using namespace std;

//Point with weight value. Structure

struct Point3D{
	float x;
	float y; 
	float z;
};

struct PointW3D{
    float x;
    float y; 
    float z;
    float w;
};

struct Node{
    Point3D nodepos; //Position of the node
    int len;		// Number of points in the node
    PointW3D *elements;	// Points in the node
};


void open_files(string name_file, int pts, PointW3D *datos){
    /* Opens the daya files. Receives the file location, number of points to read and the array of points where the data is stored */
    ifstream file;

    string mypathto_files = "../../../fake_DATA/DATOS/";
    //This creates the full path to where I have my data files
    name_file.insert(0,mypathto_files);

    file.open(name_file.c_str(), ios::in | ios::binary); //Tells the program this is a binary file using ios::binary
    if (file.fail()){
        cout << "Failed to load the file in " << name_file << endl;
        exit(1);
    }

    for ( int c = 0; c < pts; c++) //Reads line by line and stores each c line in the c PointW3D element of the array
    {
        file >> datos[c].x >> datos[c].y >> datos[c].z >> datos[c].w; 
    }
    file.close();
}

//====================================================================

void save_histogram(string name, int bns, double *histo){
    /* This function saves a one dimensional histogram in a file.
    Receives the name of the file, number of bins in the histogram and the histogram array
    */

    ofstream file2;
    file2.open(name.c_str(), ios::out | ios::binary);

    if (file2.fail()){
        cout << "Failed to save the the histogram in " << name << endl;
        exit(1);
    }
    for (int i = 0; i < bns; i++){
        file2 << histo[i] << endl;
    }
    file2.close();
}

//=================================================================== 
void add(PointW3D *&array, int &lon, float _x, float _y, float _z, float _w){
    /*
    This function manages adding points to an specific Node. It receives the previous array, longitude and point to add
    and updates the previous array and length with the same array with the new point at the end and adds +1 to the length +1

    It manages the memory allocation and free of the previous and new elements.
    */
    lon++;
    PointW3D *array_aux;
    hipMallocManaged(&array_aux, lon*sizeof(PointW3D)); 
    for (int i=0; i<lon-1; i++){
        array_aux[i].x = array[i].x;
        array_aux[i].y = array[i].y;
        array_aux[i].z = array[i].z;
        array_aux[i].w = array[i].w;
    }

    hipFree(array);
    array = array_aux;
    array[lon-1].x = _x;
    array[lon-1].y = _y;
    array[lon-1].z = _z;
    array[lon-1].w = _w;
}

void make_nodos(Node ***nod, PointW3D *dat, unsigned int partitions, float size_node, unsigned int np){
    /*
    This function classifies the data in the nodes

    Args
    nod: Node 3D array where the data will be classified
    dat: array of PointW3D data to be classified and stored in the nodes
    partitions: number nodes in each direction
    size_node: dimensions of a single node
    np: number of points in the dat array
    */

    int row, col, mom;

    // First allocate memory as an empty node:
    for (row=0; row<partitions; row++){
        for (col=0; col<partitions; col++){
            for (mom=0; mom<partitions; mom++){
                nod[row][col][mom].nodepos.z = ((float)(mom)*(size_node));
                nod[row][col][mom].nodepos.y = ((float)(col)*(size_node));
                nod[row][col][mom].nodepos.x = ((float)(row)*(size_node));
                nod[row][col][mom].len = 0;
                hipMallocManaged(&nod[row][col][mom].elements, sizeof(PointW3D));
            }
        }
    }

    // Classificate the ith elment of the data into a node and add that point to the node with the add function:
    for (int i=0; i<np; i++){
        row = (int)(dat[i].x/size_node);
        col = (int)(dat[i].y/size_node);
        mom = (int)(dat[i].z/size_node);
        add(nod[row][col][mom].elements, nod[row][col][mom].len, dat[i].x, dat[i].y, dat[i].z, dat[i].w);
    }
}

//====================================================================
//============ Kernels Section ======================================= 
//====================================================================

__global__ void count_distances11(float *XX, PointW3D *elements, int len, float ds, float dd_max, int sum){
    /*
    This global function counts the distances betweeen points within one node.

    Args:
    XX: The histogram where the distances are counted in
    elements:  Array of PointW3D points inside the node
    len: lenght of the elements array
    ds: number of bins divided by the maximum distance. Used to calculate the bin it should be counted at
    dd_max: The maximum distance of interest.
    */
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx<len-1){

        int bin;
        float d, v;
        float x1 = elements[idx].x, y1 = elements[idx].y, z1 = elements[idx].z, w1 = elements[idx].w;
        float x2,y2,z2,w2;

        for (int j=idx+1; j<len; ++j){
            x2 = elements[j].x;
            y2 = elements[j].y;
            z2 = elements[j].z;
            w2 = elements[j].w;
            d = (x2-x1)*(x2-x1)+(y2-y1)*(y2-y1)+(z2-z1)*(z2-z1);
            if (d<=dd_max+1){
                bin = (int)(sqrt(d)*ds);
                v = sum*w1*w2;
                atomicAdd(&XX[bin],v);
            }
        }
    }
}

__device__ void count_distances12(float *XX, PointW3D *elements1, int len1, PointW3D *elements2, int len2, float ds, float dd_max, int sum){
    /*
    This device function counts the distances betweeen points between two different nodes.

    Args:
    XX: The histogram where the distances are counted in
    elements1:  Array of PointW3D points inside the first node
    len1: lenght of the first elements array
    elements2:  Array of PointW3D points inside the second node
    len2: lenght of the second elements array
    ds: number of bins divided by the maximum distance. Used to calculate the bin it should be counted at
    dd_max: The maximum distance of interest.
    */

    int bin;
    float d, v;
    float x1,y1,z1,w1,x2,y2,z2,w2;

    for (int i=0; i<len1; ++i){
        x1 = elements1[i].x;
        y1 = elements1[i].y;
        z1 = elements1[i].z;
        w1 = elements1[i].w;
        for (int j=0; j<len2; ++j){
            x2 = elements2[j].x;
            y2 = elements2[j].y;
            z2 = elements2[j].z;
            w2 = elements2[j].w;
            d = (x2-x1)*(x2-x1)+(y2-y1)*(y2-y1)+(z2-z1)*(z2-z1);
            if (d<=dd_max+1){
                bin = (int)(sqrt(d)*ds);
                v = sum*w1*w2;
                atomicAdd(&XX[bin],v);
            }
        }
    }
}

__global__ void Z_direction(float *XX, Node ***nodeD, int partitions, float dd_max_node, float ds, float dd_max, int row, int col, int mom){

    int idz = (mom + 1) + blockIdx.x * blockDim.x + threadIdx.x;

    if (idz<partitions){
        float dz_nod12 = nodeD[row][col][idz].nodepos.z - nodeD[row][col][mom].nodepos.z;
        float dd_nod12 = dz_nod12*dz_nod12;
        if (dd_nod12 <= dd_max_node && nodeD[row][col][idz].len>0){
            count_distances12(XX, nodeD[row][col][mom].elements, nodeD[row][col][mom].len, nodeD[row][col][idz].elements, nodeD[row][col][idz].len, ds, dd_max, 2);
        }

    }

}

__global__ void YZ_direction_child1(float *XX, Node ***nodeD, int partitions, float dd_max_node, float ds, float dd_max, int row, int col, int mom, int idy, float dy_nod12){
    int idz = blockIdx.x * blockDim.x + threadIdx.x;
    if (idz<partitions){
        float dz_nod12 = nodeD[row][idy][idz].nodepos.z - nodeD[row][col][mom].nodepos.z;
        dz_nod12*=dz_nod12;
        float dd_nod12 = dz_nod12 + dy_nod12;
        if (dd_nod12 <= dd_max_node && nodeD[row][idy][idz].len>0){
            count_distances12(XX, nodeD[row][col][mom].elements, nodeD[row][col][mom].len, nodeD[row][idy][idz].elements, nodeD[row][idy][idz].len, ds, dd_max, 2);
        }
    }
}

__global__ void YZ_direction(float *XX, Node ***nodeD, int partitions, float dd_max_node, float ds, float dd_max, int row, int col, int mom){

    int idy = (col + 1) + blockIdx.x * blockDim.x + threadIdx.x;

    if (idy<partitions){
        float dy_nod12 = nodeD[row][idy][mom].nodepos.y - nodeD[row][col][mom].nodepos.y;
        dy_nod12*=dy_nod12;

        if (dy_nod12 <= dd_max_node){
            int blocks = (int)(ceilf((float)(partitions)/32.0));
            dim3 grid(blocks,1,1);
            dim3 block(32,1,1);
            YZ_direction_child1<<<grid,block>>>(XX, nodeD, partitions, dd_max_node, ds, dd_max, row, col, mom, idy, dy_nod12);
        }

    }

}

__global__ void XYZ_direction_child1(float *XX, Node ***nodeD, int partitions, float dd_max_node, float ds, float dd_max, int row, int col, int mom, int idx, float dx_nod12){
    int idz = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.x * blockDim.x + threadIdx.y;
    if (idz<partitions && idy<partitions){
        float dz_nod12 = nodeD[idx][idy][idz].nodepos.z - nodeD[row][col][mom].nodepos.z;
        dz_nod12*=dz_nod12;
        float dy_nod12 = nodeD[idx][idy][idz].nodepos.y - nodeD[row][col][mom].nodepos.y;
        dy_nod12*=dy_nod12;
        float dd_nod12 = dz_nod12 + dy_nod12 + dx_nod12;
        if (dd_nod12 <= dd_max_node && nodeD[idx][idy][idz].len>0){
            count_distances12(XX, nodeD[row][col][mom].elements, nodeD[row][col][mom].len, nodeD[idx][idy][idz].elements, nodeD[idx][idy][idz].len, ds, dd_max, 2);
        }
    }
}

__global__ void XYZ_direction(float *XX, Node ***nodeD, int partitions, float dd_max_node, float ds, float dd_max, int row, int col, int mom){

    int idx = (row + 1) + blockIdx.x * blockDim.x + threadIdx.x;

    if (idx<partitions){
        float dx_nod12 = nodeD[idx][col][mom].nodepos.x - nodeD[row][col][mom].nodepos.x;
        dx_nod12*=dx_nod12;

        if (dx_nod12 <= dd_max_node){
            int blocks = (int)(ceilf((float)(partitions)/32.0));
            dim3 grid(blocks,1,1);
            dim3 block(32,32,1);
            XYZ_direction_child1<<<grid,block>>>(XX, nodeD, partitions, dd_max_node, ds, dd_max, row, col, mom, idx, dx_nod12);
        }

    }

}

__global__ void make_histoXX(float *XX, Node ***nodeD, int partitions, int bn, float dmax, float size_node, int start_at){
    //If start at is 0 it does every even index, it does every odd index otherwise
    int idx = 2*(blockIdx.x * blockDim.x + threadIdx.x) + start_at;
    if (idx<(partitions*partitions*partitions)){
        //Get the node positon of this thread
        int mom = (int) (idx/(partitions*partitions));
        int col = (int) ((idx%(partitions*partitions))/partitions);
        int row = idx%partitions;
        
        if (nodeD[row][col][mom].len > 0){

            float ds = ((float)(bn))/dmax, dd_max=dmax*dmax;
            float dd_max_node = dmax + size_node*sqrt(3.0);
            dd_max_node*=dd_max_node;
            
            // Counts distances within the same node
            int blocks = (int)(ceilf((float)(nodeD[row][col][mom].len)/32.0));
            count_distances11<<<blocks,32>>>(XX, nodeD[row][col][mom].elements, nodeD[row][col][mom].len, ds, dd_max, 2);

            //Second node mobil in Z direction
            blocks = (int)(ceilf((float)(partitions-1)/32.0));
            Z_direction<<<blocks,32>>>(XX, nodeD, partitions, dd_max_node, ds, dd_max, row, col, mom);

            //Second node mobil in YZ
            YZ_direction<<<blocks,32>>>(XX, nodeD, partitions, dd_max_node, ds, dd_max, row, col, mom);

            //Second node mobil in XYZ
            XYZ_direction<<<blocks,32>>>(XX, nodeD, partitions, dd_max_node, ds, dd_max, row, col, mom);
            
        }
    }
}

__global__ void make_histoXY_child(float *XY, Node ***nodeD, int partitions, float dd_max_node, float ds, float dd_max, int row, int col, int mom){
    int idz = blockIdx.x * blockDim.z + threadIdx.z;
    int idy = blockIdx.x * blockDim.y + threadIdx.y;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idz<partitions && idy<partitions && idx<partitions){
        float dz_nod12 = nodeD[idx][idy][idz].nodepos.z - nodeD[row][col][mom].nodepos.z;
        dz_nod12*=dz_nod12;
        float dy_nod12 = nodeD[idx][idy][idz].nodepos.y - nodeD[row][col][mom].nodepos.y;
        dy_nod12*=dy_nod12;
        float dx_nod12 = nodeD[idx][idy][idz].nodepos.y - nodeD[row][col][mom].nodepos.y;
        dx_nod12*=dx_nod12;
        float dd_nod12 = dz_nod12 + dy_nod12 + dx_nod12;
        if (dd_nod12 <= dd_max_node && nodeD[idx][idy][idz].len>0){
            count_distances12(XY, nodeD[row][col][mom].elements, nodeD[row][col][mom].len, nodeD[idx][idy][idz].elements, nodeD[idx][idy][idz].len, ds, dd_max, 1);
        }
    }
}

__global__ void make_histoXY(float *XY, Node ***nodeD, Node ***nodeR, int partitions, int bn, float dmax, float size_node, int start_at){
    int row, col, mom;

    row = blockIdx.x*blockDim.x + threadIdx.x;
    col = blockIdx.x*blockDim.y + threadIdx.y;
    mom = blockIdx.x*blockDim.z + threadIdx.z;
    
    //int idx = 2*(blockIdx.x * blockDim.x + threadIdx.x) + start_at;
    //if (idx<(partitions*partitions*partitions)){

    if (row<partitions && col<partitions && mom<partitions){
        //Get the node positon in this thread
        //int mom = (int) (idx/(partitions*partitions));
        //int col = (int) ((idx%(partitions*partitions))/partitions);
        //int row = idx%partitions;
        
        if (nodeD[row][col][mom].len > 0){

            float ds = ((float)(bn))/dmax, dd_max=dmax*dmax;
            float dd_max_node = dmax + size_node*sqrt(3.0);
            dd_max_node*=dd_max_node;

            //Second node mobil in XYZ
            dim3 grid(gridDim.x ,1,1);
            dim3 block(blockDim.x,blockDim.x,blockDim.x);
            make_histoXY_child<<<grid,block>>>(XY, nodeD, partitions, dd_max_node, ds, dd_max, row, col, mom);
            
        }
    }
}

int main(int argc, char **argv){
	
    unsigned int np = stoi(argv[3]), bn = stoi(argv[4]);
    float dmax = stof(argv[5]);
    float size_box = 250.0, alpha = 2.176;
    float size_node = alpha*(size_box/pow((float)(np),1/3.));
    unsigned int partitions = (int)(ceil(size_box/size_node));

    float *DD_A, *RR_A, *DR_A, *DD_B, *RR_B, *DR_B;
    double *DD, *RR, *DR;
    PointW3D *dataD;
    PointW3D *dataR;
    hipMallocManaged(&dataD, np*sizeof(PointW3D));
    hipMallocManaged(&dataR, np*sizeof(PointW3D));

    // Name of the files where the results are saved
    string nameDD = "DDiso.dat", nameRR = "RRiso.dat", nameDR = "DRiso.dat";

    // Allocate memory for the histogram as double
    // And the subhistograms as simple presision floats
    DD = new double[bn];
    RR = new double[bn];
    DR = new double[bn];
    hipMallocManaged(&DD_A, bn*sizeof(float));
    hipMallocManaged(&RR_A, bn*sizeof(float));
    hipMallocManaged(&DR_A, bn*sizeof(float));
    hipMallocManaged(&DD_B, bn*sizeof(float));
    hipMallocManaged(&RR_B, bn*sizeof(float));
    hipMallocManaged(&DR_B, bn*sizeof(float));
    
    //Initialize the histograms in 0
    for (int i = 0; i < bn; i++){
        *(DD+i) = 0;
        *(RR+i) = 0;
        *(DR+i) = 0;
        *(DD_A+i) = 0;
        *(RR_A+i) = 0;
        *(DR_A+i) = 0;
        *(DD_B+i) = 0;
        *(RR_B+i) = 0;
        *(DR_B+i) = 0;
    }
	
	// Open and read the files to store the data in the arrays
	open_files(argv[1], np, dataD);
    open_files(argv[2], np, dataR);

    //Init the nodes arrays
    Node ***nodeD;
    Node ***nodeR;
    hipMallocManaged(&nodeR, partitions*sizeof(Node**));
    hipMallocManaged(&nodeD, partitions*sizeof(Node**));
    for (int i=0; i<partitions; i++){
        hipMallocManaged(&*(nodeR+i), partitions*sizeof(Node*));
        hipMallocManaged(&*(nodeD+i), partitions*sizeof(Node*));
        for (int j=0; j<partitions; j++){
            hipMallocManaged(&*(*(nodeR+i)+j), partitions*sizeof(Node));
            hipMallocManaged(&*(*(nodeD+i)+j), partitions*sizeof(Node));
        }
    }
    
    //Classificate the data into the nodes
    make_nodos(nodeD, dataD, partitions, size_node, np);
    make_nodos(nodeR, dataR, partitions, size_node, np);

    //Get the dimensions of the GPU grid
    int threads = 512;
    int blocks = (int)(ceil((float)((partitions*partitions*partitions)/(float)(2*threads))));
    dim3 grid(blocks,1,1);
    dim3 block(threads,1,1);
    //One thread for each node

    clock_t begin = clock();
    //Launch the kernels
    make_histoXX<<<grid,block>>>(DD_A, nodeD, partitions, bn, dmax, size_node, 0);
    make_histoXX<<<grid,block>>>(DD_B, nodeD, partitions, bn, dmax, size_node, 1);
    make_histoXX<<<grid,block>>>(RR_A, nodeR, partitions, bn, dmax, size_node, 0);
    make_histoXX<<<grid,block>>>(RR_B, nodeR, partitions, bn, dmax, size_node, 1);

    blocks = (int)(ceil((float)(partitions)/8.0));
    dim3 grid_XY(blocks,1,1);
    dim3 block_XY(8,8,8);
    make_histoXY<<<grid_XY,block_XY>>>(DR_A, nodeD, nodeR, partitions, bn, dmax, size_node, 0);
    make_histoXY<<<grid_XY,block_XY>>>(DR_B, nodeD, nodeR, partitions, bn, dmax, size_node, 1);

    //Waits for the GPU to finish
    hipDeviceSynchronize();  

    //Check here for errors
    hipError_t error = hipGetLastError(); 
    cout << "The error code is " << error << endl;
    if(error != 0)
    {
      // print the CUDA error message and exit
      printf("CUDA error: %s\n", hipGetErrorString(error));
      exit(-1);
    }

    clock_t end = clock();
    double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
    printf("\nSpent time = %.4f seg.\n", time_spent );

    //Collect the subhistograms data into the double precision main histograms
    //THis has to be done in CPU since GPU only allows single precision
    for (int i = 0; i < bn; i++){
        DD[i] = (double)(DD_A[i]+ DD_B[i]);
        RR[i] = (double)(RR_A[i]+ RR_B[i]);
        DR[i] = (double)(DR_A[i]+ DR_B[i]);
    }

    cout << "Termine de hacer todos los histogramas" << endl;
    /*
    // Shows the histograms
    cout << "\nHistograma DD:" << endl;
    int sum = 0;
    for (int k = 0; k<bn; k++){
        cout << DD[k] << "\t";
        sum += DD[k];
    }
    cout << "Total: " << sum << endl;

    cout << "\nHistograma RR:" << endl;
    for (int k = 0; k<bn; k++){
        cout << RR[k] << "\t";
    }

    cout << "\nHistograma DR:" << endl;
    for (int k = 0; k<bn; k++){
        cout << DR[k] << "\t";
    }
    */
	
	// Guardamos los histogramas
	save_histogram(nameDD, bn, DD);
	cout << "Guarde histograma DD..." << endl;
	save_histogram(nameRR, bn, RR);
	cout << "Guarde histograma RR..." << endl;
	save_histogram(nameDR, bn, DR);
	cout << "Guarde histograma DR..." << endl;

    //Free the memory
    hipFree(&dataD);
    hipFree(&dataR);

    delete[] DD;
    delete[] DR;
    delete[] RR;
    hipFree(&DD_A);
    hipFree(&RR_A);
    hipFree(&DR_A);
    hipFree(&DD_B);
    hipFree(&RR_B);
    hipFree(&DR_B);


    for (int i=0; i<partitions; i++){
        for (int j=0; j<partitions; j++){
            hipFree(&*(*(nodeR+i)+j));
            hipFree(&*(*(nodeD+i)+j));
        }
        hipFree(&*(nodeR+i));
        hipFree(&*(nodeD+i));
    }
    hipFree(&nodeR);
    hipFree(&nodeD);

    cout << "Programa Terminado..." << endl;
    return 0;
}

