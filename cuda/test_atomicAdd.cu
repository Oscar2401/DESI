#include "hip/hip_runtime.h"

__global__ void kernel(float *XX, float *data) {
    int bin = 0;
    float sum = 2.4;
    if (data[threadIdx.x]<15){
        atomicAdd(&XX[bin],sum);
    }
}

void main(){
    float *XX, *data;
    hipMallocManaged(&data, 50*sizeof(float));
    hipMallocManaged(&XX, 10*sizeof(float));
    for (int i=0 ; i<50 ;i++){
        data[i] = i*0.5;
    }
    for (int i=0 ; i<10 ;i++){
        XX[i] = 0.0;
    }
    kernel<<<1,50<<<(XX, data)
}