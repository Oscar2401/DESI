#include "hip/hip_runtime.h"
#include <iostream>


__global__ void kernel(float *XX, float *data) {
    int bin = 0;
    float sum = 2.4f;
    if (data[threadIdx.x]<15){
        atomicAdd(&XX[bin],sum);
    }
}

int main(){
    float *XX;
    float *data;
    hipMallocManaged(&data, 50*sizeof(float));
    hipMallocManaged(&XX, 10*sizeof(float));
    for (int i=0 ; i<50 ;i++){
        data[i] = i*0.5;
    }
    for (int i=0 ; i<10 ;i++){
        XX[i] = 0.0;
    }
    kernel<<<1,50>>>(XX, data);

    std::cout << __CUDA_ARCH__ << std::endl;
    std::cout << XX[0] << std::endl;

    return 0;
}