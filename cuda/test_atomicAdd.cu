
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#include <stdio.h>


__global__ void kernel(float XX, float *data) {
    //int bin = 0;
    float sum = 2.4f;
    if (data[threadIdx.x]<15){
        printf("%f", sum);
        atomicAdd(&XX,sum);
        printf("%f", XX);
    }
}

int main(){
    float XX=0;
    float *data;
    hipMallocManaged(&data, 50*sizeof(float));
    //cudaMallocManaged(&XX, 10*sizeof(float));
    for (int i=0 ; i<50 ;i++){
        data[i] = i*0.5;
    }
    /*
    for (int i=0 ; i<10 ;i++){
        XX[i] = 0.0;
    }
    */
    kernel<<<1,50>>>(XX, data);

    std::cout << XX << std::endl;

    return 0;
}