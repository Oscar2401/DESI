#include "hip/hip_runtime.h"
#include <iostream>


__global__ void kernel(float XX, float *data) {
    //int bin = 0;
    float sum = 2.4f;
    if (data[threadIdx.x]<15){
        atomicAdd(&XX,sum);
    }
}

int main(){
    float XX=0;
    float *data;
    hipMallocManaged(&data, 50*sizeof(float));
    //hipMallocManaged(&XX, 10*sizeof(float));
    for (int i=0 ; i<50 ;i++){
        data[i] = i*0.5;
    }
    /*
    for (int i=0 ; i<10 ;i++){
        XX[i] = 0.0;
    }
    */
    kernel<<<1,50>>>(XX, data);

    std::cout << XX[0] << std::endl;

    return 0;
}