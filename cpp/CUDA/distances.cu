#include "hip/hip_runtime.h"
// nvcc distances.cu -o o.out && ./o.out data.dat rand0.dat 32768 30 180
#include<iostream>
#include<fstream>
#include<string.h>
#include <stdio.h>
#include <math.h>

using namespace std;

//Structura que define un punto 3D
//Accesa a cada componente con var.x, var.y, var.z
struct Punto{
    double x,y,z;
};

struct Node{
    //Punto nodepos;	// Coordenadas del nodo (posición del nodo) // Se obtiene con las coordenadas del nodo.
    //int in_vicinage;    //Cantidad de nodos vecinos.
    //int *nodes_vicinage;     // Array con los master id de localizacion de los nodos vecinos.
    int len;		// Cantidad de elementos en el nodo.
    Punto *elements;
};

void read_file(string file_loc, Punto *data){
    //cout << file_loc << endl;
    string line; //No uso esta variable realmente, pero con eof() no se detenía el loop
    
    ifstream archivo(file_loc);
    
    if (archivo.fail() | !archivo ){
        cout << "Error al cargar el archivo " << endl;
        exit(1);
    }
    
    
    int n_line = 1;
    if (archivo.is_open() && archivo.good()){
        archivo >> data[0].x >> data[0].y >> data[0].z;
        while(getline(archivo, line)){
            archivo >> data[n_line].x >> data[n_line].y >> data[n_line].z;
            n_line++;
        }
    }
    //cout << "Succesfully readed " << file_loc << endl;
}

void guardar_Histograma(string nombre,int dim, long int *histograma){
    ofstream archivo;
    archivo.open(nombre.c_str(),ios::out | ios::binary);
    if (archivo.fail()){
        cout << "Error al guardar el archivo " << endl;
        exit(1);
    }
    for (int i = 0; i < dim; i++)
    {
        archivo << histograma[i] << endl;
    }
    archivo.close();
}

float distance(Punto p1, Punto p2){
    float x = p1.x-p2.x, y=p1.y-p2.y, z=p1.z-p2.z;
    return sqrt(x*x + y*y + z*z);
}

__device__
void count_3_N111(int row, int col, int mom, long int ***XXX, Node ***nodeS){
    /*
    Funcion para contar los triangulos en un mismo Nodo.

    row, col, mom => posición del Nodo. Esto define al Nodo.

    */
    int i,j,k;
    int a,b,c;
    float dd_max = 180;
    float ds = 30.0f/180.0f;
    float dx,dy,dz;
    float d12,d13,d23;
    float x1,y1,z1,x2,y2,z2,x3,y3,z3;

    for (i=0; i<nodeS[row][col][mom].len-2; ++i){
        x1 = nodeS[row][col][mom].elements[i].x;
        y1 = nodeS[row][col][mom].elements[i].y;
        z1 = nodeS[row][col][mom].elements[i].z;
        for (j=i+1; j<nodeS[row][col][mom].len-1; ++j){
            x2 = nodeS[row][col][mom].elements[j].x;
            y2 = nodeS[row][col][mom].elements[j].y;
            z2 = nodeS[row][col][mom].elements[j].z;
            dx = x2-x1;
            dy = y2-y1;
            dz = z2-z1;
            d12 = dx*dx+dy*dy+dz*dz;
            if (d12<=dd_max){
            for (k=j+1; k<nodeS[row][col][mom].len; ++k){ 
                x3 = nodeS[row][col][mom].elements[k].x;
                y3 = nodeS[row][col][mom].elements[k].y;
                z3 = nodeS[row][col][mom].elements[k].z;
                dx = x3-x1;
                dy = y3-y1;
                dz = z3-z1;
                d13 = dx*dx+dy*dy+dz*dz;
                if (d13<=dd_max){
                dx = x3-x2;
                dy = y3-y2;
                dz = z3-z2;
                d23 = dx*dx+dy*dy+dz*dz;
                if (d23<=dd_max){
                    a = (int)(sqrt(d12)*ds);
                    b = (int)(sqrt(d13)*ds);
                    c = (int)(sqrt(d23)*ds);
                    XXX[a][b][c] = atomicAdd(&&&XXX[a][b][c],1);
                    //*(*(*(XXX+(int)(sqrt(d12)*ds))+(int)(sqrt(d13)*ds))+(int)(sqrt(d23)*ds))+=1;
                }
                }
            }
            }
        }
    }
    printf("Exiting subkernel \n");
}

// Kernel function to populate the grid of nodes
__global__
void histo_XXX(Node ***tensor_node, long int ***XXX, unsigned int partitions)
{
    if (blockIdx.x<partitions && threadIdx.x<partitions && threadIdx.y<partitions ){
        unsigned int row, col, mom;
        row = threadIdx.x;
        col = threadIdx.y;
        mom = blockIdx.x;
        count_3_N111(row, col, mom, XXX, tensor_node);
        printf("Exit the kernel \n");
    }
}

void add_neighbor(int *&array, int &lon, int id){
    lon++;
    /*
    int *array_aux;
    hipMallocManaged(&array_aux, lon*sizeof(int)); 
    for (int i=0; i<lon-1; i++){
        array_aux[i] = array[i];
    }
    hipFree(&array);
    array = array_aux;
    */
    array[lon-1] = id;
}

//=================================================================== 
void add(Punto *&array, int &lon, float _x, float _y, float _z){
    lon++;
    Punto *array_aux; // = new Punto[lon];
    hipMallocManaged(&array_aux, lon*sizeof(Punto)); 
    for (int i=0; i<lon-1; i++){
        array_aux[i].x = array[i].x;
        array_aux[i].y = array[i].y;
        array_aux[i].z = array[i].z;
    }
    hipFree(&array);
    array = array_aux;
    array[lon-1].x = _x;
    array[lon-1].y = _y; 
    array[lon-1].z = _z; 
}

void make_nodos(Node ***nod, Punto *dat, unsigned int partitions, float size_node, unsigned int n_pts, float d_max){
    /*
    Función para crear los nodos con los datos y puntos random

    Argumentos
    nod: arreglo donde se crean los nodos.
    dat: datos a dividir en nodos.

    */
    int test = 0;
    int row, col, mom, node_id, id_max = pow((int) d_max/size_node + 1,2);
    int n_row, n_col, n_mom, internodal_distance; // Row, Col and Mom of the possible node in the neighborhood

    // Inicializamos los nodos vacíos:
    cout << "Initialize empty nodes" << endl;
    for (row=0; row<partitions; row++){
        for (col=0; col<partitions; col++){
            for (mom=0; mom<partitions; mom++){

                nod[row][col][mom].len = 0;
                hipMallocManaged(&nod[row][col][mom].elements, sizeof(Punto));
            }
        }
    }
    cout << "The nodes have 0 elements each and 0 neighbors" << endl;

    // Llenamos los nodos con los puntos de dat:
    cout << "Started the classification" << endl;
    for (int i=0; i<n_pts; i++){
        row = (int)(dat[i].x/size_node);
        col = (int)(dat[i].y/size_node);
        mom = (int)(dat[i].z/size_node);
        add(nod[row][col][mom].elements, nod[row][col][mom].len, dat[i].x, dat[i].y, dat[i].z);
    }
    cout << "Finished the classification" << endl;
}

int main(int argc, char **argv){
        
    string data_loc = argv[1];
    string rand_loc = argv[2];
    string mypathto_files = "../../fake_DATA/DATOS/";
    //This creates the full path to where I have my data files
    data_loc.insert(0,mypathto_files);
    rand_loc.insert(0,mypathto_files);
    
    unsigned int n_pts = stoi(argv[3]), bn=stoi(argv[4]);
    unsigned int n_even = n_pts+(n_pts%2!=0);
    float d_max=stof(argv[5]), size_box = 250.0, size_node = 2.17*size_box/bn;
    unsigned int partitions = (int)(ceil(size_box/size_node));
    double dbin = d_max/(double)bn;
    
    // Crea los histogramas
    //cout << "Histograms initialization" << endl;
    long int ***DDD;
    // inicializamos los histogramas
    hipMallocManaged(&DDD, bn*sizeof(long int**));
    for (int i=0; i<bn; i++){
        hipMallocManaged(&*(DDD+i), bn*sizeof(long int*));
        for (int j = 0; j < bn; j++){
            hipMallocManaged(&*(*(DDD+i)+j), bn*sizeof(long int));
        }
    }
    //Inicializa en 0
    for (int i=0; i<bn; i++){
        for (int j=0; j<bn; j++){
            for (int k = 0; k < bn; k++){
                DDD[i][j][k]= 0;
            }
        }
    }
    //cout << "Finished histograms initialization" << endl;

    //cout << "Starting to read the data files" << endl;
    Punto *data, *rand; //Crea un array de n_pts puntos
    hipMallocManaged(&data, n_pts*sizeof(Punto));
    hipMallocManaged(&rand, n_pts*sizeof(Punto));
    //Llama a una funcion que lee los puntos y los guarda en la memoria asignada a data y rand
    read_file(data_loc,data);
    read_file(rand_loc,rand);
    cout << "Successfully readed the data" << endl;

    //Create Nodes
    //cout << "Started nodes initialization" << endl;
    Node ***nodeD;
    hipMallocManaged(&nodeD, partitions*sizeof(Node**));
    for (int i=0; i<partitions; i++){
        hipMallocManaged(&*(nodeD+i), partitions*sizeof(Node*));
        for (int j=0; j<partitions; j++){
            hipMallocManaged(&*(*(nodeD+i)+j), partitions*sizeof(Node));
        }
    }
    //cout << "Finished nodes initialization" << endl;
    //cout << "Started the data classification into the nodes." << endl;
    make_nodos(nodeD, data, partitions, size_node, n_pts, d_max);
    cout << "Finished the data classification in node" << endl;

    //cout << "Calculating the nuber of blocks and threads for the kernel for XXX" << endl;
    //Sets GPU arrange of threads
    int threads=1, blocks=n_even, threads_test, blocks_test;
    float score=pow(blocks,2)+pow((blocks*threads)-n_even,2), score_test;
    for (int i=1; i<6; i++){
        threads_test = pow(2,i);
        blocks_test = (int)(n_even/threads_test)+1;
        score_test = pow(blocks_test,2)+pow((blocks_test*threads_test)-n_even,2);
        
        if (score_test<score){
            threads=threads_test;
            blocks=blocks_test;
            score=score_test;
        }
    }
    
    cout << "Entering to the kernel" << endl;
    dim3 grid(16,1,1);
    dim3 block(16,16);
    histo_XXX<<<grid,block>>>(nodeD, DDD, partitions);

    //Waits for the GPU to finish
    hipDeviceSynchronize();

    cout << DDD[7][7][7] << endl;

    // Free memory
    // Free the histogram arrays
    cout << "Free the histograms allocated memory" << endl;
    for (int i=0; i<bn; i++){
        for (int j = 0; j < bn; j++){
            hipFree(&*(*(DDD+i)+j));
        }
        hipFree(&*(DDD+i));
    }
    hipFree(&DDD);
    //Free the nodes and their inner arrays.
    cout << "Free the nodes allocated memory" << endl;
    for (int i=0; i<partitions; i++){
        for (int j=0; j<partitions; j++){
            hipFree(&*(*(nodeD+i)+j));
        }
        hipFree(&*(nodeD+i));
    }
    hipFree(&nodeD);
    //Free data and random arrays
    cout << "Free the data allocated memory" << endl;
    hipFree(&data);
    hipFree(&rand);

    cout << "Finished the program" << endl;

    return 0;
}
