#include "hip/hip_runtime.h"

#include<iostream>
#include<fstream>
#include<vector>
#include<string.h>

#include <stdio.h>
#include <math.h>

using namespace std;

//Structura que define un punto 3D
//Accesa a cada componente con var.x, var.y, var.z
struct Punto{
    double x,y,z;
};

struct Node{
    Punto nodepos;	// Coordenadas del nodo (posición del nodo)
    int len=0;		// Cantidad de elementos en el nodo.
    Punto *elements;
};

void read_file(string file_loc, Punto *data){
    //cout << file_loc << endl;
    string line; //No uso esta variable realmente, pero con eof() no se detenía el loop
    
    ifstream archivo(file_loc);
    
    if (archivo.fail() | !archivo ){
        cout << "Error al cargar el archivo " << endl;
        exit(1);
    }
    
    
    int n_line = 1;
    if (archivo.is_open() && archivo.good()){
        archivo >> data[0].x >> data[0].y >> data[0].z;
        while(getline(archivo, line)){
            archivo >> data[n_line].x >> data[n_line].y >> data[n_line].z;
            n_line++;
        }
    }
    //cout << "Succesfully readed " << file_loc << endl;
}

void guardar_Histograma(string nombre,int dim, long int *histograma){
    ofstream archivo;
    archivo.open(nombre.c_str(),ios::out | ios::binary);
    if (archivo.fail()){
        cout << "Error al guardar el archivo " << endl;
        exit(1);
    }
    for (int i = 0; i < dim; i++)
    {
        archivo << histograma[i] << endl;
    }
    archivo.close();
}

float distance(Punto p1, Punto p2){
    float x = p1.x-p2.x, y=p1.y-p2.y, z=p1.z-p2.z;
    return sqrt(x*x + y*y + z*z);
}

__global__
void XY(float *dest, float *a, float *b, int *N){
    int p_id = threadIdx.x + blockDim.x*blockIdx.x;
    int id = threadIdx.y + blockDim.y*blockIdx.y;

    if (id < *N && p_id <*N){
        int x = id*3;
        int y = x+1;
        int z = y+1;

        int p_x = p_id*3;
        int p_y = p_x+1;
        int p_z = p_y+1;
        float d;
        //float histo[30];
        int bin;
        d = sqrt(pow(a[p_x] - b[x],2)+pow(a[p_y]-b[y],2) + pow(a[p_z]-b[z],2));
        if (d<=180){
            bin = (int) (d/6.0);
            atomicAdd(&dest[bin],1);
        }
    }
}

__global__
void XX(float *dest, float *a, int *N){
    int p_id = threadIdx.x + blockDim.x*blockIdx.x;
    int id = threadIdx.y + blockDim.y*blockIdx.y;

    if (p_id<*N && id<*N && p_id<id){

        int p_x = p_id*3;
        int p_y = p_x+1;
        int p_z = p_y+1;

        float d;
        int bin;

        int x = id*3;
        int y = x+1;
        int z = y+1;

        d = sqrt(pow(a[p_x] - a[x],2)+pow(a[p_y]-a[y],2) + pow(a[p_z]-a[z],2));
        if (d<=180){
            bin = (int) (d/6.0);
            atomicAdd(&dest[bin],2);
        }
    }
}

// Kernel function to populate the grid of nodes
__global__
void create_grid(double *test, Punto *datos, unsigned int n_pts)
{
    if (blockIdx.x==0 && blockIdx.y==0 && blockIdx.y==0 && threadIdx.x==0 && threadIdx.y==0 && threadIdx.z==0 ){
       //printf("%i \n", threadIdx.x);
       test = datos[1].x + datos[1].y +datos[1].z;
    }
    
    /*
    for(int i=0; i<n_pts;i++){
        nodeid = (int)(datos[i].x/size_node) + (int)((datos[i].y/size_node))*partitions + (int)((datos[i].z/size_node))*partitions*partitions;
        //node_grid[nodeid].elements[node_grid[nodeid].len]=datos[i];
        node_grid[nodeid].len++;
        printf("El valor es %d.\n", *(node_grid[nodeid].elements+1));
    }
    */
}

int main(int argc, char **argv){
        
    string data_loc = argv[1];
    string rand_loc = argv[2];
    string mypathto_files = "../../fake_DATA/DATOS/";
    //This creates the full path to where I have my data files
    data_loc.insert(0,mypathto_files);
    rand_loc.insert(0,mypathto_files);
    
    unsigned int n_pts = stoi(argv[3]), bn=stoi(argv[4]);
    unsigned int N_even = n_pts+(n_pts%2!=0);
    float d_max=stof(argv[5]), size_box = 250.0, size_node = 2.17*size_box/bn;
    unsigned int partitions = (int)(ceil(size_box/size_node));
    double dbin = d_max/(double)bn;
    
    // Crea los histogramas
    long int ***DDD, ***DDR, ***DRR, ***RRR;
    // inicializamos los histogramas
    DDD = new long int**[bn];
    RRR = new long int**[bn];
    DDR = new long int**[bn];
    DRR = new long int**[bn];

    for (int i=0; i<bn; i++){
        *(DDD+i) = new long int*[bn];
        *(RRR+i) = new long int*[bn];
        *(DDR+i) = new long int*[bn];
        *(DRR+i) = new long int*[bn];
        for (int j = 0; j < bn; j++){
            *(*(DDD+i)+j) = new long int[bn];
            *(*(RRR+i)+j) = new long int[bn];
            *(*(DDR+i)+j) = new long int[bn];
            *(*(DRR+i)+j) = new long int[bn];
        }
    }
    
    //Inicializa en 0
    for (int i=0; i<bn; i++){
        for (int j=0; j<bn; j++){
            for (int k = 0; k < bn; k++){
                DDD[i][j][k]= 0;
                DDR[i][j][k]= 0;   
                DRR[i][j][k]= 0;
                RRR[i][j][k]= 0;
            }
        }
    }

    Punto *data = new Punto[n_pts]; //Crea un array de n_pts puntos
    Punto *rand = new Punto[n_pts]; //Crea un array de N puntos

    //Punto *data, *rand; //, *d_data, *d_rand;
    hipMallocManaged(&data, n_pts*sizeof(Punto));
    hipMallocManaged(&rand, n_pts*sizeof(Punto));

    //Llama a una funcion que lee los puntos y los guarda en la memoria asignada a data y rand
    read_file(data_loc,data);
    read_file(rand_loc,rand);

    //Sets GPU arrange of threads
    int threads=1, blocks=N_even, threads_test, blocks_test;
    float score=pow(blocks,2)+pow((blocks*threads)-N_even,2), score_test;
    for (int i=1; i<6; i++){
        threads_test = pow(2,i);
        blocks_test = (int)(N_even/threads_test)+1;
        score_test = pow(blocks_test,2)+pow((blocks_test*threads_test)-N_even,2);
        
        if (score_test<score){
            threads=threads_test;
            blocks=blocks_test;
            score=score_test;
        }
    }


    double *test;
    test = 0;
    hipMallocManaged(&test, sizeof(double));
    create_grid<<<1,256>>>(test, data, n_pts);

    //Waits for the GPU to finish
    hipDeviceSynchronize();

    cout << test-(double)24.909824 << endl;

    // Free memory
    hipFree(&test);
    hipFree(&data);
    hipFree(&rand);

    return 0;
}
