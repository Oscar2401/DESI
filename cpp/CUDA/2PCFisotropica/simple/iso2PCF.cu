
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream> //manejo de archivos
#include <string.h>
#include <time.h>
#include <math.h>

using namespace std;

struct Point3D{
    float x;
    float y;
    float z;
};

//====================================================================
//============ Sección de Funciones ================================== 
//====================================================================

void open_files(string name_file, int pts, Point3D *datos){
    /* Función para abrir nuestros archivos de datos */
    ifstream file;

    string mypathto_files = "../../../../fake_DATA/DATOS/";
    //This creates the full path to where I have my data files
    name_file.insert(0,mypathto_files);

    file.open(name_file.c_str(), ios::in | ios::binary); //le indico al programa que se trata de un archivo binario con ios::binary
    if (file.fail()){
        cout << "Error al cargar el archivo " << endl;
        exit(1);
    }

    //int c=0,remove;
    int remove;
    //while (!file.eof())
    for ( int c = 0; c < pts; c++)
    {
        file >> datos[c].x >> datos[c].y >> datos[c].z >> remove; 
        //c++;
    }
    file.close();
}

//====================================================================

void save_histogram(string name, int bns, unsigned int *histo){
    /* Función para guardar nuestros archivos de histogramas */
    ofstream file2;
    file2.open(name.c_str(), ios::out | ios::binary);

    if (file2.fail()){
        cout << "Error al guardar el archivo " << endl;
        exit(1);
    }
    for (int i = 0; i < bns; i++){
        file2 << histo[i] << endl;
    }
    file2.close();
}

// Métodos para hacer histogramas.
__global__ void make_histoXX(unsigned int *XX, Point3D *data, int n_pts, float ds, float dd_max){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx<n_pts-1){
        printf("%f \n",  blockDim.x);
        int pos; // Posición de apuntador.
        float dis, dx, dy, dz;
        for(int j = idx+1; j < n_pts; j++){
            dx = data[idx].x-data[j].x;
            dy = data[idx].y-data[j].y;
            dz = data[idx].z-data[j].z;
            dis = dx*dx + dy*dy + dz*dz;
            //printf("%f \n", dis);
            //printf("%f \n", sqrt(dis));
            //printf("%f \n", sqrt(dis)*ds);

            if(dis <= dd_max){
                pos = (int)(sqrt(dis)*ds);
                atomicAdd(&XX[pos],2);
            }
        }
    }
}
__global__ void make_histoXY(unsigned int *XY, Point3D *dataD, Point3D *dataR, int n_pts, float ds, float dd_max){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx<n_pts-1){
        int pos;
        float dis, dx, dy, dz;
        for(int j = 0; j < n_pts; j++){
            dx = dataD[idx].x-dataR[j].x;
            dy = dataD[idx].y-dataR[j].y;
            dz = dataD[idx].z-dataR[j].z;
            dis = dx*dx + dy*dy + dz*dz;
            if(dis <= dd_max){
                pos = (int)(sqrt(dis)*ds);
                atomicAdd(&XY[pos],1);
            }
        }
    }
}

int main(int argc, char **argv){
	
    int np = stoi(argv[3]), bn = stoi(argv[4]);
    float dmax = stof(argv[5]);
    float ds = (float)(bn)/dmax, dd_max=dmax*dmax;
    //int np = 32768, bn = 10;
    //float dmax = 180.0;

    unsigned int *DD, *RR, *DR;
    Point3D *dataD;
    Point3D *dataR;
    hipMallocManaged(&dataD, np*sizeof(Point3D));// Asignamos meoria a esta variable
    hipMallocManaged(&dataR, np*sizeof(Point3D));

    // Nombre de los archivos 
    string nameDD = "DDiso.dat", nameRR = "RRiso.dat", nameDR = "DRiso.dat";
    /*
    nameDD.append(argv[3]);
    nameRR.append(argv[3]);
    nameDR.append(argv[3]);
    nameDD += ".dat";
    nameRR += ".dat";
    nameDR += ".dat";
    */

    // Creamos los histogramas
    hipMallocManaged(&DD, bn*sizeof(unsigned int));
    hipMallocManaged(&RR, bn*sizeof(unsigned int));
    hipMallocManaged(&DR, bn*sizeof(unsigned int));
    
    for (int i = 0; i < bn; i++){
        *(DD+i) = 0.0; // vector[i]
        *(RR+i) = 0.0;
        *(DR+i) = 0.0;
    }
	
	// Abrimos y trabajamos los datos en los histogramas
	open_files(argv[1], np, dataD);
    open_files(argv[2], np, dataR); // guardo los datos en los Struct
    
    dim3 grid((int)(ceil((float)(np/(float)(1024))),1,1));
    dim3 block(1024,1,1);

    clock_t begin = clock();
    cout << ds << endl;
    make_histoXX<<<grid,block>>>(DD, dataD, np, ds, dd_max);
    make_histoXX<<<grid,block>>>(RR, dataR, np, ds, dd_max);
    make_histoXY<<<grid,block>>>(DR, dataD, dataR, np, ds, dd_max);

    //Waits for the GPU to finish
    hipDeviceSynchronize();  

    //Check here for errors
    hipError_t error = hipGetLastError(); 
    cout << "The error code is " << error << endl;
    if(error != 0)
    {
      // print the CUDA error message and exit
      printf("CUDA error: %s\n", hipGetErrorString(error));
      exit(-1);
    }

    clock_t end = clock();
    double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
    printf("\nTiempo en CPU usado = %.4f seg.\n", time_spent );
    
	cout << "Termine de hacer todos los histogramas" << endl;
	// Mostramos los histogramas 
	cout << "\nHistograma DD:" << endl;
	for (int k = 0; k<bn; k++){
		printf("%d \t",DD[k]);
	}
	cout << "\nHistograma RR:" << endl;
	for (int k = 0; k<bn; k++){
		printf("%d \t",RR[k]);
	}
	cout << "\nHistograma DR:" << endl;
	for (int k = 0; k<bn; k++){
		printf("%d \t",DR[k]);
	}
	
	
	// Guardamos los histogramas
	save_histogram(nameDD, bn, DD);
	cout << "Guarde histograma DD..." << endl;
	save_histogram(nameRR, bn, RR);
	cout << "Guarde histograma RR..." << endl;
	save_histogram(nameDR, bn, DR);
	cout << "Guarde histograma DR..." << endl;

    hipFree(&dataD);
    hipFree(&dataR);
    hipFree(&DD);
    hipFree(&RR);
    hipFree(&DR);

    cout << "Programa Terminado..." << endl;
    return 0;
}

