
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
// Kernel function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
    y[i] = x[i] + y[i];
}

int main(void)
{
  int N = 1<<20;
  float *x, *y;

  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // Run kernel on 1M elements on the GPU
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    dim3 grid(numBlocks,1,1);
    dim3 block(blockSize,1,1);
    add<<<grid, block>>>(N, x, y);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

    // check for error
    hipError_t error = hipGetLastError();
    std::cout << error << std::endl;
    if(error != 0)
    {
    // print the CUDA error message and exit
    printf("CUDA error: %s\n", hipGetErrorString(error));
    exit(-1);
    }

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  hipFree(x);
  hipFree(y);
  
  return 0;
}